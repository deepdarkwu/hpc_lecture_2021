
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <chrono>
using namespace std;

__global__ void cuda_matmul(float *subA, float *subB, float *subC, int N) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0;
  extern __shared__ float S[];
  
  for (int ks=0; ks<N; ks+=blockDim.x) {
    __syncthreads();
    S[threadIdx.x] = subA[N*i+ks+threadIdx.x];
    __syncthreads();
    for (int k=ks; k<ks+blockDim.x; k++) {
      sum +=S[k-ks] * subB[N*k+j];
    }
  }
   atomicAdd(&subC[N*i+j], sum);
}

int main(int argc, char **argv) {
  
  int N = 2048;
  int M = 256;
  
  float *subA;
  float *subB;
  float *subC;
  hipMallocManaged(&subA, N * N * sizeof(float));
  hipMallocManaged(&subB, N * N * sizeof(float));
  hipMallocManaged(&subC, N * N * sizeof(float));
  
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      subA[N*i+j] = drand48();
      subB[N*i+j] = drand48();
      subC[N*i+j] = 0;
    }
  }
  
  
  auto tic = chrono::steady_clock::now();
  
  dim3 grid(N/M, N);
  cuda_matmul<<<grid,M,M*sizeof(float)>>>(subA, subB, subC, N);
  hipDeviceSynchronize();
  
  auto toc = chrono::steady_clock::now();
  double comp_time = chrono::duration<double>(toc - tic).count();


  for (int i=0; i<N; i++)
    for (int k=0; k<N; k++)
      for (int j=0; j<N; j++)
        subC[N*i+j] -= subA[N*i+k] * subB[N*k+j];
  
  double err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(subC[N*i+j]);
  
  
  printf("N    : %d\n",N);
  printf("comp : %lf s\n", comp_time);
  printf("total: %lf s (%lf GFlops)\n", comp_time, 2.*N*N*N/comp_time/1e9);
  printf("error: %lf\n",err/N/N);
  
  hipFree(subA);
  hipFree(subB);
  hipFree(subC);
}
